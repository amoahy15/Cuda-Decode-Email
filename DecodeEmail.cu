#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024

__global__ void decrypt(char* data, int size);

int main(int argc, char** argv) {
   
    if (argc != 2) {
        printf("Error: Incorrect number of arguments.\n");
        return 1;
    }

   
    FILE* file = fopen(argv[1], "r");
    if (!file) {
        printf("File does not exist %s\n", argv[1]);
        return 1;
    }
    fseek(file, 0, SEEK_END);
    int size = ftell(file);
    fseek(file, 0, SEEK_SET);

 
    char* hostData = (char*)malloc(size);
    char* deviceData;
    hipMalloc((void**)&deviceData, size);

    
    fread(hostData, 1, size, file);

    hipMemcpy(deviceData, hostData, size, hipMemcpyHostToDevice);

    int blocks = (size + N - 1) / N;
    decrypt<<<blocks, N>>>(deviceData, size);


    hipMemcpy(hostData, deviceData, size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize(); 

    printf("Decoded data:\n%.*s", size, hostData);

    fclose(file);
    free(hostData);
    hipFree(deviceData);

    exit(0);
}
__global__ void decrypt(char* data, int size) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < size) {
        data[i] -= 1;
    }
}